
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>
using namespace std;

void random_ints(int* a, int num)
{
	for (int i = 0; i < num; ++i){
		a[i] = rand() % 20;
		// cout<<a[i]<<"\t";
	}
	// cout<<endl<<endl;
}


//Calculate on the kernel
__global__ void add(int* a, int* b, int* c)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}
	
int main(int argc, char const *argv[])
{
	int scale=10, blocks = 1, threads_per_block= 1;
	if(argc == 3){
		scale = atoi(argv[1]); 
		threads_per_block = atoi(argv[2]);
	}
	blocks = ceil((double)scale/threads_per_block);

	cout<<"--------------------\n";
	cout<<"Scale:\t"<< scale << endl;
	cout<<"Blocks:\t" << blocks << endl;
	cout<<"threads per block:\t" << threads_per_block << endl;
	cout<<"----------"<<endl;

	srand(time(NULL));

	// clock_t begin, end;
	clock_t gpu_begin, gpu_end;
	// double time_spent;
	double gpu_time_spent;


	//host copy
	int *a,*b,*c;

	//device copy 
	int *d_a,*d_b,*d_c;

	int size = scale * sizeof(int);

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	// for (int i=0; i<scale; ++i){
	// 	c[i] = 0;
	// }
	random_ints(a, scale);
	random_ints(b, scale);


	//allocate GPU space
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);


	//copy input to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// cudaMemcpy(d_c, c, size, cudaMemcpyHostToDevice);
	gpu_begin = clock();

	add<<<blocks,threads_per_block>>>(d_a, d_b, d_c);

	gpu_end = clock();
	gpu_time_spent = (double)(gpu_end - gpu_begin)* 1000 / CLOCKS_PER_SEC;



	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	// for (int i=0; i<scale; ++i){
	// 	cout<< c[i]<<"\t";
	// }
	// cout<<endl<<endl<<endl;

	//kernel
	// cout<<"c is "<<c<<endl;
	cout<<"time_spent "<<gpu_time_spent<<"ms"<<endl;
	cout<<"--------------------\n\n";

	free(a);
	free(b);
	free(c);

	return 0;
}